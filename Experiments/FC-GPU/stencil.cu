#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <csignal>
#include <thread>
#include <future>
#include <fstream>
#include <vector>
#include <chrono>
#include <iomanip>
#include <sys/ipc.h>
#include <sys/shm.h>
#include <unistd.h>
#include "shared_data.h"

using namespace std;
using steady_clock = std::chrono::steady_clock;

float setpoint, period_ms, termination_ms;
const int JOBS = 1;
bool workload_changed = false;
bool createdshm = false;
typedef double Real;
Real *h_Vm, *h_dVm, *h_sigma;   // host views (pinned, mapped)
Real *d_Vm, *d_dVm, *d_sigma;   // GPU aliases returned by hipHostGetDevicePointer
#ifndef SIGNAL_TYPE
    #ifdef T1
        #define SIGNAL_TYPE SIGUSR1
        #define SHARED_MEM_INDEX 0
        #define SHARED_MEM_INDEX2 0
        #define LOG_SUFFIX "1"
    #elif defined(T2)
        #define SIGNAL_TYPE SIGUSR2
        #define SHARED_MEM_INDEX 1
        #define SHARED_MEM_INDEX2 1
        #define LOG_SUFFIX "2"
    #else
        #define SIGNAL_TYPE SIGUSR1
        #define SHARED_MEM_INDEX 0
        #define SHARED_MEM_INDEX2 0
        #define LOG_SUFFIX "1"
    #endif
#endif

#define MAKE_LOG_PATH(prefix) ("logs/" prefix LOG_SUFFIX ".txt")

ofstream outputfile(MAKE_LOG_PATH("log"));
ofstream rtj1(MAKE_LOG_PATH("rtj"));
ofstream period1(MAKE_LOG_PATH("p"));
ofstream preemptiontime(MAKE_LOG_PATH("preemptiont"), ios::app);

#define BSIZE 16
#define XTILE 20

const int size = 512;
int repeat = 100;
const int nx = size, ny = size, nz = size;
const int vol = nx * ny * nz;



steady_clock::time_point start_time, next_release, preemptionlaunch;


__global__ void stencil3d(
    const Real *__restrict__ d_psi, 
          Real *__restrict__ d_npsi, 
    const Real *__restrict__ d_sigmaX, 
    const Real *__restrict__ d_sigmaY, 
    const Real *__restrict__ d_sigmaZ,
    int nx, int ny, int nz)
{
    __shared__ Real sm_psi[4][BSIZE][BSIZE];

    const int tjj = threadIdx.y;
    const int tkk = threadIdx.x;

    #define V0(y,z) sm_psi[pii][y][z]
    #define V1(y,z) sm_psi[cii][y][z]
    #define V2(y,z) sm_psi[nii][y][z]

    #define sigmaX(x,y,z,dir) d_sigmaX[ z + nz * ( y + ny * ( x + nx * dir ) ) ]
    #define sigmaY(x,y,z,dir) d_sigmaY[ z + nz * ( y + ny * ( x + nx * dir ) ) ]
    #define sigmaZ(x,y,z,dir) d_sigmaZ[ z + nz * ( y + ny * ( x + nx * dir ) ) ]

    #define psi(x,y,z) d_psi[ z + nz * ( y + ny * ( x ) ) ]
    #define npsi(x,y,z) d_npsi[ z + nz * ( y + ny * ( x ) ) ]

    d_psi = &(psi(XTILE * blockIdx.x, (BSIZE - 2) * blockIdx.y, (BSIZE - 2) * blockIdx.z));
    d_npsi = &(npsi(XTILE * blockIdx.x, (BSIZE - 2) * blockIdx.y, (BSIZE - 2) * blockIdx.z));

    int nLast_x = XTILE + 1;
    int nLast_y = (BSIZE - 1);
    int nLast_z = (BSIZE - 1);

    if (blockIdx.x == gridDim.x - 1) nLast_x = nx - 2 - XTILE * blockIdx.x + 1;
    if (blockIdx.y == gridDim.y - 1) nLast_y = ny - 2 - (BSIZE - 2) * blockIdx.y + 1;
    if (blockIdx.z == gridDim.z - 1) nLast_z = nz - 2 - (BSIZE - 2) * blockIdx.z + 1;

    if (tjj > nLast_y || tkk > nLast_z) return;

    int pii = 0, cii = 1, nii = 2, tii;
    sm_psi[cii][tjj][tkk] = psi(0, tjj, tkk);
    sm_psi[nii][tjj][tkk] = psi(1, tjj, tkk);
    Real xcharge, ycharge, zcharge, dV = 0;

    __syncthreads();

    for (int ii = 1; ii < nLast_x; ii++) {
        sm_psi[nii][tjj][tkk] = psi(ii + 1, tjj, tkk);
        __syncthreads();

        // Compute and accumulate charges here (omitted for brevity)

        __syncthreads();
        tii = pii;
        pii = cii;
        cii = nii;
        nii = tii;
    }
}


void allocateAndInitMemory()
{
    // page-locked host memory that the GPU can directly address
    hipSetDeviceFlags(hipDeviceMapHost);        

    hipHostAlloc(&h_Vm,    sizeof(Real) * vol,     hipHostMallocMapped);
    hipHostAlloc(&h_sigma, sizeof(Real) * vol * 9, hipHostMallocMapped);
    hipHostAlloc(&h_dVm,   sizeof(Real) * vol,     hipHostMallocMapped);

    // obtain the matching device pointers (aliases)
    hipHostGetDevicePointer((void**)&d_Vm,    h_Vm,    0);
    hipHostGetDevicePointer((void**)&d_sigma, h_sigma, 0);
    hipHostGetDevicePointer((void**)&d_dVm,   h_dVm,   0);

    // initialise on host – no hipMemcpy needed
    for (int i = 0; i < vol;      ++i) h_Vm[i]    = i % 19;
    for (int i = 0; i < vol * 9;  ++i) h_sigma[i] = i % 19;
    hipMemset(h_dVm, 0, sizeof(Real) * vol);  // still fine: h_dVm is host
}

void freeMemory()
{
    hipHostFree(h_Vm);
    hipHostFree(h_sigma);
    hipHostFree(h_dVm);   // releases both host & device views
}

void kernellaunch(dim3 gridDim, dim3 blockDim) {
    auto pre = steady_clock::now();
    float pre_ms = std::chrono::duration<float, std::milli>(pre - preemptionlaunch).count();
    preemptiontime << "Preemption=" << std::fixed << std::setprecision(3) << pre_ms << " ms\n";

    float total_gpu = 0;
    for (int j = 0; j < JOBS; ++j) {
        hipEvent_t st, sp;
        hipEventCreate(&st); hipEventCreate(&sp);
        hipEventRecord(st);
        for (int r = 0; r < repeat; ++r) {
            stencil3d<<<gridDim, blockDim>>>(d_Vm, d_dVm, d_sigma, d_sigma + 3 * vol, d_sigma + 6 * vol, nx, ny, nz);
        }
        hipEventRecord(sp); hipEventSynchronize(sp);
        float el;
        hipEventElapsedTime(&el, st, sp);
        total_gpu += el;
        hipEventDestroy(st); hipEventDestroy(sp);
    }

    sharedData->values[SHARED_MEM_INDEX] = pre_ms + total_gpu / JOBS;
    sharedData->executiontime[SHARED_MEM_INDEX] = total_gpu / JOBS;
}

void runPeriodicService() {
    allocateAndInitMemory();
    int bdimx = (nx - 2) / XTILE + ((nx - 2) % XTILE != 0);
    int bdimy = (ny - 2) / (BSIZE - 2) + ((ny - 2) % (BSIZE - 2) != 0);
    int bdimz = (nz - 2) / (BSIZE - 2) + ((nz - 2) % (BSIZE - 2) != 0);

    dim3 gridDim(bdimx, bdimy, bdimz);
    dim3 blockDim(BSIZE, BSIZE, 1);
    sharedData->newperiods[SHARED_MEM_INDEX] = period_ms;

    start_time = steady_clock::now();
    next_release = start_time;
    const float change_trigger = termination_ms / 2.0f;

    while (true) {
        std::this_thread::sleep_until(next_release);
        auto now = steady_clock::now();
        float elapsed_ms = std::chrono::duration<float, std::milli>(now - start_time).count();
        if (elapsed_ms >= termination_ms) break;

   

        preemptionlaunch = steady_clock::now();
        future<void> fut = async(launch::async, [&]() {
            kernellaunch(gridDim, blockDim);
        });
        fut.get();

        period1 << period_ms << '\n';
        rtj1 << sharedData->values[SHARED_MEM_INDEX] << '\n';
        next_release += std::chrono::milliseconds((long)(sharedData->newperiods[SHARED_MEM_INDEX]));
    }

    freeMemory();
}

int main(int argc, char* argv[]) {
    if (argc < 4) {
        cerr << "Usage: ./stencil <setpoint> <period(ms)> <termination(s)>\n";
        return 1;
    }

    setpoint = atof(argv[1]);
    period_ms = atof(argv[2]);
    termination_ms = atof(argv[3]) * 1000.0f;

    key_t key = ftok("shmfile", 65);
    int shmid = shmget(key, 1024, 0666 | IPC_CREAT);
    if (shmid > 0) {
        sharedData = (SharedData*)shmat(shmid, nullptr, 0);
        createdshm = true;
    } else {
        shmid = shmget(key, sizeof(SharedData), 0666);
        sharedData = (SharedData*)shmat(shmid, nullptr, 0);
    }

    cout << "[T" << SHARED_MEM_INDEX << "] Real-Time Stencil Task Started.\n";
    runPeriodicService();

    if (createdshm) {
        shmdt(sharedData);
        shmctl(shmid, IPC_RMID, NULL);
    }

    outputfile.close(); rtj1.close(); period1.close(); preemptiontime.close();
    return 0;
}
